/*!
 * Copyright 2018 XGBoost contributors
 */
#include "common.h"

namespace xgboost {
namespace common {

int AllVisibleGPUs() {
  int n_visgpus = 0;
  try {
    // When compiled with CUDA but running on CPU only device,
    // hipGetDeviceCount will fail.
    hipGetDeviceCount(&n_visgpus);
  } catch(const dmlc::Error &except) {
    return 0;
  } catch(const std::exception& e) {
    return 0;
  } catch(const std::string& e) {
    return 0;
  } catch(...) {
    return 0;
  }
  return n_visgpus;
}

}  // namespace common
}  // namespace xgboost
